#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>
#include <algorithm>
#include <random>
#include <cmath>
#include <fstream>
#include <limits>

#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/generate.h>
#include <thrust/copy.h>
#include <thrust/random.h>
#include <thrust/transform.h>
// #include <thrust/iterator/counting_iterator.h>
#include <thrust/functional.h>
#include <thrust/for_each.h>
#include <thrust/sequence.h>


// Dimensions of surfaces (lengths)
constexpr double X = 1;
constexpr double Y = 1;
constexpr double Z = 2;

// Number of trials and points
constexpr int trials = (int)5e3;
constexpr int N = (int)1e5;

double sdCalc(float mean, thrust::host_vector<float> data);

void writeCSV(thrust::host_vector<float> data);

struct integral_functor : public thrust::unary_function<float,float>
{
    __host__ __device__
    float operator()(int the_seed) {

        // seed a random number generator
        thrust::default_random_engine rng(the_seed);
        // create a mapping from random numbers to [0,1)
        thrust::uniform_real_distribution<float> dist(0,1);

        float x, yh, theta, phi, dy, yv, z;
        int hits = 0;

        for (int i=0; i < N; i++) {
            // random values on a horizontal surface
            x = dist(rng) * X;
            yh = dist(rng) * Y;
            theta = acosf(1 - 2 * dist(rng)) / 2.0;
            phi = dist(rng) * M_PI - M_PI/2.0;

            // points on a vertical plane
            dy = x * tanf(phi);
            yv = yh + dy;
            z = sqrtf(x*x + dy*dy) * tanf(M_PI/2.0 - theta);

            // see if there's a hit
            if (z > 0 && z < Z && yv > 0 && yv < Y)
                hits += 1;
        }
 
        return ((float)hits)/(2.0 * (float)N);
    }
};

int main() {

    thrust::host_vector<int> rvec(trials);
    thrust::device_vector<float> vf_dev(trials);

    // make seeds
    thrust::default_random_engine rng(time(NULL));
    thrust::uniform_int_distribution<int> dist {0, std::numeric_limits<int>::max()};
    thrust::generate(rvec.begin(), rvec.end(), [&] { return dist(rng); });

    // copy to device and perform MC
    thrust::device_vector<float> rvec_dev = rvec;
    thrust::transform(rvec_dev.begin(), rvec_dev.end(), vf_dev.begin(), integral_functor());

    // copy back to host
    thrust::host_vector<float> vf = vf_dev;

    float mu = thrust::reduce(vf.begin(), vf.end()) / ((float)vf.size());
    double sigma = sdCalc(mu, vf);
    double se = sigma / sqrtf((float) trials);

    std::cout << "mu " << mu << "\nsigma " << sigma << "\nse " << se << std::endl;

    writeCSV(vf);
    
}

// standard deviation
double sdCalc(float mean, thrust::host_vector<float> data) {
    double ret = 0.0;
    for (int i=0; i < data.size(); i++)
        ret += pow(data[i] - mean, 2);
    
    return sqrt(ret / ((double) data.size()));
}

void writeCSV(thrust::host_vector<float> data) {
    std::ofstream outfile("./out.csv");
    for (int i=0; i < data.size(); i++) {
        outfile << i << ", " << data[i] << "\n";
    }
    outfile.close();
    std::ofstream vffile("./vf.csv");
    for (int i=0; i < data.size(); i++) {
        vffile << data[i] << "\n";
    }
    vffile.close();
}